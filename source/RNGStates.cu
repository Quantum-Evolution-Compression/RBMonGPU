#include "hip/hip_runtime.h"
#include "RNGStates.hpp"
#include "types.h"
#include <iostream>


using namespace std;


namespace rbm_on_gpu {

namespace kernel {

__global__ void initialize_random_states(hiprandState_t* random_states, const unsigned int num_states) {
    const auto idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < num_states) {
        hiprand_init(0, idx, 0u, &random_states[idx]);
    }
}

}


RNGStates::RNGStates(const unsigned int num_states, const bool gpu)
:
    num_states(num_states),
    rng_states_device(nullptr),
    rng_states_host(nullptr),
    gpu(gpu) {

    if(this->gpu) {
        CUDA_CHECK(hipMalloc(&this->rng_states_device, sizeof(hiprandState_t) * this->num_states));

        const auto blockDim = 256u;
        kernel::initialize_random_states<<<this->num_states / blockDim + 1u, blockDim>>>(
            this->rng_states_device,
            this->num_states
        );
    }
    else {
        this->rng_states_host = new mt19937[num_states];
        for(auto i = 0u; i < this->num_states; i++) {
            this->rng_states_host[i] = mt19937(i);
        }
    }
}

RNGStates::~RNGStates() noexcept(false) {
    if(this->gpu) {
        CUDA_FREE(this->rng_states_device);
    }
    else {
        delete[] this->rng_states_host;
    }
}


}
