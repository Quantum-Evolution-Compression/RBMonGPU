#include "hip/hip_runtime.h"
#include "network_functions/HilbertSpaceDistance.hpp"
#include "spin_ensembles.hpp"
#include "quantum_states.hpp"

#include <cstring>
#include <math.h>


namespace rbm_on_gpu {

namespace kernel {


template<bool compute_gradient, bool free_quantum_axis, typename Psi_t, typename Psi_t_prime, typename SpinEnsemble>
void kernel::HilbertSpaceDistance::compute_averages(
    const Psi_t& psi, const Psi_t_prime& psi_prime, const Operator& operator_,
    const bool is_unitary, const SpinEnsemble& spin_ensemble
) const {
    const auto num_params = psi_prime.get_num_params();

    const auto this_ = *this;
    const auto psi_kernel = psi.get_kernel();
    const auto psi_prime_kernel = psi_prime.get_kernel();
    const auto N = psi.get_num_spins();

    spin_ensemble.foreach(
        psi,
        [=] __device__ __host__ (
            const unsigned int spin_index,
            const Spins spins,
            const complex_t log_psi,
            typename Psi_t::Angles& angles,
            const double weight
        ) {
            #include "cuda_kernel_defines.h"

            SHARED complex_t local_energy;
            operator_.local_energy(local_energy, psi_kernel, spins, log_psi, angles);

            SHARED typename Psi_t_prime::Angles angles_prime;
            angles_prime.init(psi_prime_kernel, spins);

            SHARED complex_t log_psi_prime;
            psi_prime_kernel.log_psi_s(log_psi_prime, spins, angles_prime);

            SHARED complex_t psi_i_ratio[MAX_SPINS];
            if(free_quantum_axis) {
                SHARED Spins spins_i;
                SHARED complex_t log_psi_prime_i;
                for(auto i = 0u; i < N; i++) {
                    SYNC;
                    SINGLE {
                        spins_i = spins.flip(i);
                    }
                    SYNC;

                    angles_prime.init(psi_prime_kernel, spins_i);
                    psi_prime_kernel.log_psi_s(log_psi_prime_i, spins_i, angles_prime);
                    SINGLE {
                        psi_i_ratio[i] = exp(log_psi_prime_i - log_psi_prime);
                    }
                }
                SYNC;

                MULTI(i, N) {
                    generic_atomicAdd(
                        &log_psi_prime,
                        (
                            this_.delta_alpha[i] * spins[i] * psi_i_ratio[i] +
                            this_.delta_beta[i] * complex_t(0.0, 1.0) * (
                                psi_i_ratio[i] * this_.cos_sum_alpha[i] -
                                spins[i] * this_.sin_sum_alpha[i]
                            )
                        )
                    );
                }
                SYNC;

                // TODO: optimize
                angles_prime.init(psi_prime_kernel, spins);
            }

            SHARED complex_t   omega;
            SHARED double      probability_ratio;

            SINGLE
            {
                if(is_unitary) {
                    omega = exp(conj(log_psi_prime - log_psi)) * local_energy;
                    generic_atomicAdd(
                        this_.next_state_norm_avg,
                        weight * (local_energy * conj(local_energy)).real()
                    );
                }
                else {
                    omega = exp(local_energy + conj(log_psi_prime - log_psi));
                    generic_atomicAdd(
                        this_.next_state_norm_avg,
                        weight * exp(2 * local_energy.real())
                    );
                }
                probability_ratio = exp(2.0 * (log_psi_prime.real() - log_psi.real()));

                generic_atomicAdd(this_.omega_avg, weight * omega);
                generic_atomicAdd(this_.probability_ratio_avg, weight * probability_ratio);
            }

            if(compute_gradient) {
                if(free_quantum_axis) {
                    MULTI(i, N) {
                        const auto O_alpha_i = spins[i] * psi_i_ratio[i];
                        generic_atomicAdd(&this_.omega_O_k_avg[i], weight * omega * conj(O_alpha_i));
                        generic_atomicAdd(&this_.probability_ratio_O_k_avg[i], weight * probability_ratio * conj(O_alpha_i));

                        const auto O_beta_i = complex_t(0.0, 1.0) * (
                            psi_i_ratio[i] * this_.cos_sum_alpha[i] -
                            spins[i] * this_.sin_sum_alpha[i]
                        );
                        generic_atomicAdd(&this_.omega_O_k_avg[N + i], weight * omega * conj(O_beta_i));
                        generic_atomicAdd(&this_.probability_ratio_O_k_avg[N + i], weight * probability_ratio * conj(O_beta_i));
                    }
                }

                psi_prime_kernel.foreach_O_k(
                    spins,
                    angles_prime,
                    [&](const unsigned int k, const complex_t& O_k_element) {
                        generic_atomicAdd(&this_.omega_O_k_avg[k], weight * omega * conj(O_k_element));
                        generic_atomicAdd(&this_.probability_ratio_O_k_avg[k], weight * probability_ratio * conj(O_k_element));
                    }
                );
            }
        },
        max(psi.get_width(), psi_prime.get_width())
    );
}

} // namespace kernel

HilbertSpaceDistance::HilbertSpaceDistance(const unsigned int N, const unsigned int num_params, const bool gpu)
      : num_params(num_params),
        omega_avg_ar(1, gpu),
        omega_O_k_avg_ar(num_params, gpu),
        probability_ratio_avg_ar(1, gpu),
        probability_ratio_O_k_avg_ar(num_params, gpu),
        next_state_norm_avg_ar(1, gpu),
        delta_alpha_ar(N, gpu),
        delta_beta_ar(N, gpu),
        sin_sum_alpha_ar(N, gpu),
        cos_sum_alpha_ar(N, gpu) {
    this->gpu = gpu;

    this->omega_avg = this->omega_avg_ar.data();
    this->omega_O_k_avg = this->omega_O_k_avg_ar.data();
    this->probability_ratio_avg = this->probability_ratio_avg_ar.data();
    this->probability_ratio_O_k_avg = this->probability_ratio_O_k_avg_ar.data();
    this->next_state_norm_avg = this->next_state_norm_avg_ar.data();

    this->delta_alpha = this->delta_alpha_ar.data();
    this->delta_beta = this->delta_beta_ar.data();
    this->sin_sum_alpha = this->sin_sum_alpha_ar.data();
    this->cos_sum_alpha = this->cos_sum_alpha_ar.data();
}

template<typename Psi_t, typename Psi_t_prime>
void HilbertSpaceDistance::update_quaxis(const Psi_t& psi, const Psi_t_prime& psi_prime) {
    for(auto i = 0u; i < psi.get_num_spins(); i++) {
        const auto delta_alpha = psi_prime.alpha_array[i] - psi.alpha_array[i];
        const auto sum_alpha = psi_prime.alpha_array[i] + psi.alpha_array[i];
        const auto delta_beta = psi_prime.beta_array[i] - psi.beta_array[i];
        this->delta_alpha_ar[i] = delta_alpha;
        this->delta_beta_ar[i] = delta_beta;
        this->sin_sum_alpha_ar[i] = sin(sum_alpha);
        this->cos_sum_alpha_ar[i] = cos(sum_alpha);
    }

    this->delta_alpha_ar.update_device();
    this->delta_beta_ar.update_device();
    this->sin_sum_alpha_ar.update_device();
    this->cos_sum_alpha_ar.update_device();
}


void HilbertSpaceDistance::clear() {
    this->omega_avg_ar.clear();
    this->omega_O_k_avg_ar.clear();
    this->probability_ratio_avg_ar.clear();
    this->probability_ratio_O_k_avg_ar.clear();
    this->next_state_norm_avg_ar.clear();
}


template<typename Psi_t, typename Psi_t_prime, typename SpinEnsemble>
double HilbertSpaceDistance::distance(
    const Psi_t& psi, const Psi_t_prime& psi_prime, const Operator& operator_, const bool is_unitary,
    const SpinEnsemble& spin_ensemble
) {
    this->clear();
    if(psi.free_quantum_axis) {
        this->update_quaxis(psi, psi_prime);
        this->compute_averages<false, true>(psi, psi_prime, operator_, is_unitary, spin_ensemble);
    }
    else {
        this->compute_averages<false, false>(psi, psi_prime, operator_, is_unitary, spin_ensemble);
    }

    this->omega_avg_ar.update_host();
    this->probability_ratio_avg_ar.update_host();
    this->next_state_norm_avg_ar.update_host();

    this->omega_avg_ar.front() /= spin_ensemble.get_num_steps();
    this->probability_ratio_avg_ar.front() /= spin_ensemble.get_num_steps();
    this->next_state_norm_avg_ar.front() /= spin_ensemble.get_num_steps();

    // return this->probability_ratio_avg_ar.front();
    return sqrt(
        1.0 - (this->omega_avg_ar.front() * conj(this->omega_avg_ar.front())).real() / (
            this->next_state_norm_avg_ar.front() *this->probability_ratio_avg_ar.front()
        )
    );
}


template<typename Psi_t, typename Psi_t_prime, typename SpinEnsemble>
double HilbertSpaceDistance::gradient(
    complex<double>* result, const Psi_t& psi, const Psi_t_prime& psi_prime, const Operator& operator_,
    const bool is_unitary, const SpinEnsemble& spin_ensemble
) {
    this->clear();
    if(psi.free_quantum_axis) {
        this->update_quaxis(psi, psi_prime);
        this->compute_averages<true, true>(psi, psi_prime, operator_, is_unitary, spin_ensemble);
    }
    else {
        this->compute_averages<true, false>(psi, psi_prime, operator_, is_unitary, spin_ensemble);
    }

    this->omega_avg_ar.update_host();
    this->omega_O_k_avg_ar.update_host();
    this->probability_ratio_avg_ar.update_host();
    this->probability_ratio_O_k_avg_ar.update_host();
    this->next_state_norm_avg_ar.update_host();

    this->omega_avg_ar.front() /= spin_ensemble.get_num_steps();
    this->probability_ratio_avg_ar.front() /= spin_ensemble.get_num_steps();
    this->next_state_norm_avg_ar.front() /= spin_ensemble.get_num_steps();

    const auto u = (this->omega_avg_ar.front() * conj(this->omega_avg_ar.front())).real();
    const auto v = this->next_state_norm_avg_ar.front() * this->probability_ratio_avg_ar.front();
    const auto distance = sqrt(1.0 - u / v);

    for(auto k = 0u; k < this->num_params; k++) {
        this->omega_O_k_avg_ar.at(k) *= 1.0 / spin_ensemble.get_num_steps();
        this->probability_ratio_O_k_avg_ar.at(k) *= 1.0 / spin_ensemble.get_num_steps();

        const auto u_k_prime = conj(this->omega_avg_ar.front()) * this->omega_O_k_avg_ar[k];
        const auto v_k_prime = this->next_state_norm_avg_ar.front() * this->probability_ratio_O_k_avg_ar[k];

        result[k] = (
            -0.5 * (u_k_prime * v - u * v_k_prime) / (v * v)
        ).to_std() / distance;

        // result[k] = 2.0 * v_k_prime.to_std();
    }

    return distance;
    // return v;
}


#ifdef ENABLE_MONTE_CARLO

#ifdef ENABLE_PSI

template double HilbertSpaceDistance::distance(const Psi& psi, const Psi& psi_prime, const Operator& operator_, const bool is_unitary, const MonteCarloLoop& spin_ensemble);
template double HilbertSpaceDistance::gradient(complex<double>* result, const Psi& psi, const Psi& psi_prime, const Operator& operator_, const bool is_unitary, const MonteCarloLoop& spin_ensemble);

#endif

#ifdef ENABLE_PSI_DEEP

template double HilbertSpaceDistance::distance(const PsiDeep& psi, const PsiDeep& psi_prime, const Operator& operator_, const bool is_unitary, const MonteCarloLoop& spin_ensemble);
template double HilbertSpaceDistance::gradient(complex<double>* result, const PsiDeep& psi, const PsiDeep& psi_prime, const Operator& operator_, const bool is_unitary, const MonteCarloLoop& spin_ensemble);

#endif

#ifdef ENABLE_PSI_PAIR

template double HilbertSpaceDistance::distance(const PsiPair& psi, const PsiPair& psi_prime, const Operator& operator_, const bool is_unitary, const MonteCarloLoop& spin_ensemble);
template double HilbertSpaceDistance::gradient(complex<double>* result, const PsiPair& psi, const PsiPair& psi_prime, const Operator& operator_, const bool is_unitary, const MonteCarloLoop& spin_ensemble);

#endif

#if defined(ENABLE_PSI_CLASSICAL) && defined(ENABLE_PSI_DEEP)

template double HilbertSpaceDistance::distance(const PsiClassical& psi, const PsiDeep& psi_prime, const Operator& operator_, const bool is_unitary, const MonteCarloLoop& spin_ensemble);
template double HilbertSpaceDistance::gradient(complex<double>* result, const PsiClassical& psi, const PsiDeep& psi_prime, const Operator& operator_, const bool is_unitary, const MonteCarloLoop& spin_ensemble);

#endif

#if defined(ENABLE_PSI_CLASSICAL) && defined(ENABLE_PSI_PAIR)

template double HilbertSpaceDistance::distance(const PsiClassical& psi, const PsiPair& psi_prime, const Operator& operator_, const bool is_unitary, const MonteCarloLoop& spin_ensemble);
template double HilbertSpaceDistance::gradient(complex<double>* result, const PsiClassical& psi, const PsiPair& psi_prime, const Operator& operator_, const bool is_unitary, const MonteCarloLoop& spin_ensemble);

#endif


#endif



#ifdef ENABLE_EXACT_SUMMATION

#ifdef ENABLE_PSI

template double HilbertSpaceDistance::distance(const Psi& psi, const Psi& psi_prime, const Operator& operator_, const bool is_unitary, const ExactSummation& spin_ensemble);
template double HilbertSpaceDistance::gradient(complex<double>* result, const Psi& psi, const Psi& psi_prime, const Operator& operator_, const bool is_unitary, const ExactSummation& spin_ensemble);

#endif

#ifdef ENABLE_PSI_DEEP

template double HilbertSpaceDistance::distance(const PsiDeep& psi, const PsiDeep& psi_prime, const Operator& operator_, const bool is_unitary, const ExactSummation& spin_ensemble);
template double HilbertSpaceDistance::gradient(complex<double>* result, const PsiDeep& psi, const PsiDeep& psi_prime, const Operator& operator_, const bool is_unitary, const ExactSummation& spin_ensemble);

#endif

#ifdef ENABLE_PSI_PAIR

template double HilbertSpaceDistance::distance(const PsiPair& psi, const PsiPair& psi_prime, const Operator& operator_, const bool is_unitary, const ExactSummation& spin_ensemble);
template double HilbertSpaceDistance::gradient(complex<double>* result, const PsiPair& psi, const PsiPair& psi_prime, const Operator& operator_, const bool is_unitary, const ExactSummation& spin_ensemble);

#endif

#if defined(ENABLE_PSI_CLASSICAL) && defined(ENABLE_PSI_DEEP)

template double HilbertSpaceDistance::distance(const PsiClassical& psi, const PsiDeep& psi_prime, const Operator& operator_, const bool is_unitary, const ExactSummation& spin_ensemble);
template double HilbertSpaceDistance::gradient(complex<double>* result, const PsiClassical& psi, const PsiDeep& psi_prime, const Operator& operator_, const bool is_unitary, const ExactSummation& spin_ensemble);

#endif

#if defined(ENABLE_PSI_CLASSICAL) && defined(ENABLE_PSI_PAIR)

template double HilbertSpaceDistance::distance(const PsiClassical& psi, const PsiPair& psi_prime, const Operator& operator_, const bool is_unitary, const ExactSummation& spin_ensemble);
template double HilbertSpaceDistance::gradient(complex<double>* result, const PsiClassical& psi, const PsiPair& psi_prime, const Operator& operator_, const bool is_unitary, const ExactSummation& spin_ensemble);

#endif


#endif


} // namespace rbm_on_gpu
