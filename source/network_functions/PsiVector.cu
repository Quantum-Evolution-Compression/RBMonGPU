
#include <hip/hip_runtime.h>
#ifdef ENABLE_EXACT_SUMMATION


#include "network_functions/PsiVector.hpp"
#include "quantum_states.hpp"
#include "spin_ensembles/ExactSummation.hpp"
#include "types.h"


namespace rbm_on_gpu {


template<typename Psi_t>
void psi_vector(complex<double>* result, const Psi_t& psi) {
    ExactSummation exact_summation(psi.get_num_spins(), psi.gpu);

    complex_t* result_ptr;
    MALLOC(result_ptr, sizeof(complex_t) * exact_summation.get_num_steps(), psi.gpu);

    const auto log_prefactor = log(psi.prefactor);
    auto psi_kernel = psi.get_kernel();

    exact_summation.foreach(
        psi,
        [=] __host__ __device__ (
            const unsigned int spin_index,
            const Spins spins,
            const complex_t log_psi,
            const typename Psi_t::Angles& angles,
            const double weight
        ) {
            #ifdef __CUDA_ARCH__
            if(threadIdx.x == 0)
            #endif
            {
                result_ptr[spin_index] = exp(log_prefactor + log_psi);
            }
        }
    );

    MEMCPY_TO_HOST(result, result_ptr, sizeof(complex_t) * exact_summation.get_num_steps(), psi.gpu);
    FREE(result_ptr, psi.gpu);
}

template<typename Psi_t>
Array<complex_t> psi_vector(const Psi_t& psi) {
    Array<complex_t> result(1 << psi.get_num_spins(), false);
    psi_vector(reinterpret_cast<complex<double>*>(result.data()), psi);

    return result;
}


#ifdef ENABLE_PSI
template void psi_vector(complex<double>* result, const Psi& psi);
template Array<complex_t> psi_vector(const Psi& psi);
#endif // ENABLE_PSI

#ifdef ENABLE_PSI_DEEP
template void psi_vector(complex<double>* result, const PsiDeep& psi);
template Array<complex_t> psi_vector(const PsiDeep& psi);
#endif // ENABLE_PSI_DEEP

#ifdef ENABLE_PSI_PAIR
template void psi_vector(complex<double>* result, const PsiPair& psi);
template Array<complex_t> psi_vector(const PsiPair& psi);
#endif // ENABLE_PSI_PAIR

#ifdef ENABLE_PSI_CLASSICAL
template void psi_vector(complex<double>* result, const PsiClassical& psi);
template Array<complex_t> psi_vector(const PsiClassical& psi);
#endif // ENABLE_PSI_CLASSICAL

#ifdef ENABLE_PSI_EXACT
template void psi_vector(complex<double>* result, const PsiExact& psi);
template Array<complex_t> psi_vector(const PsiExact& psi);
#endif // ENABLE_PSI_EXACT

} // namespace rbm_on_gpu


#endif // ENABLE_EXACT_SUMMATION
