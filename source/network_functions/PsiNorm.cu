#include "hip/hip_runtime.h"
#include "network_functions/PsiNorm.hpp"
#include "quantum_state/Psi.hpp"
#include "quantum_state/PsiDeep.hpp"
#include "quantum_state/PsiPair.hpp"
#include "spin_ensembles/ExactSummation.hpp"
#include "types.h"

namespace rbm_on_gpu {


template<typename Psi_t>
double psi_norm(const Psi_t& psi, const ExactSummation& exact_summation) {
    double* result_ptr;
    MALLOC(result_ptr, sizeof(double), psi.gpu);
    MEMSET(result_ptr, 0, sizeof(double), psi.gpu);

    auto this_ = psi.get_kernel();

    exact_summation.foreach(
        psi,
        [=] __host__ __device__ (
            const unsigned int spin_index,
            const Spins spins,
            const complex_t log_psi,
            const typename Psi_t::Angles& angles,
            const double weight
        ) {
            #ifdef __CUDA_ARCH__
            if(threadIdx.x == 0)
            #endif
            {
                generic_atomicAdd(result_ptr, this_.probability_s(log_psi.real()));
            }
        }
    );

    double result;
    MEMCPY_TO_HOST(&result, result_ptr, sizeof(double), psi.gpu);
    FREE(result_ptr, psi.gpu);

    return sqrt(result);
}


template double psi_norm(const Psi& psi, const ExactSummation&);
template double psi_norm(const PsiDeep& psi, const ExactSummation&);
template double psi_norm(const PsiPair& psi, const ExactSummation&);

} // namespace rbm_on_gpu
