
#include <hip/hip_runtime.h>
#ifdef ENABLE_PSI_DEEP

#include "quantum_state/PsiDeep.hpp"

#include <complex>
#include <vector>
#include <random>
#include <cstring>
#include <algorithm>
#include <iterator>


namespace rbm_on_gpu {

template<typename dtype>
PsiDeepT<dtype>::PsiDeepT(const PsiDeepT<dtype>& other)
    :
    alpha_array(other.alpha_array),
    beta_array(other.beta_array),
    layers(other.layers),
    free_quantum_axis(other.free_quantum_axis),
    gpu(other.gpu)
{
    this->N = other.N;
    this->prefactor = other.prefactor;
    this->num_layers = other.num_layers;
    this->width = other.width;
    this->num_units = other.num_units;

    this->init_kernel();
}


template<typename dtype>
void PsiDeepT<dtype>::init_kernel() {
    this->num_params = 2 * this->N; // alpha and beta
    auto angle_idx = 0u;
    for(auto layer_idx = 0u; layer_idx < this->num_layers; layer_idx++) {
        const auto& layer = *next(this->layers.begin(), layer_idx);
        auto& kernel_layer = kernel::PsiDeepT<dtype>::layers[layer_idx];
        kernel_layer.size = layer.size;
        kernel_layer.lhs_connectivity = layer.lhs_connectivity;

        kernel_layer.begin_params = this->num_params;
        kernel_layer.begin_angles = angle_idx;

        this->num_params += layer.size + layer.lhs_weights.size();
        angle_idx += layer.size;
    }
    for(auto layer_idx = 0u; layer_idx < this->num_layers; layer_idx++) {
        auto& layer = kernel::PsiDeepT<dtype>::layers[layer_idx];
        auto next_layer = kernel::PsiDeepT<dtype>::layers + layer_idx + 1;

        layer.rhs_connectivity = (
            layer_idx + 1 < this->num_layers ?
            next_layer->size * next_layer->lhs_connectivity / layer.size :
            0u
        );
    }
    this->O_k_length = this->num_params - 2 * this->N;

    this->update_kernel();
}


template<typename dtype>
void PsiDeepT<dtype>::update_kernel() {
    for(auto layer_idx = 0u; layer_idx < this->num_layers; layer_idx++) {
        Layer& layer = *next(this->layers.begin(), layer_idx);
        auto& kernel_layer = kernel::PsiDeepT<dtype>::layers[layer_idx];

        kernel_layer.lhs_connections = layer.lhs_connections.data();
        kernel_layer.rhs_connections = layer.rhs_connections.data();
        kernel_layer.lhs_weights = layer.lhs_weights.data();
        kernel_layer.rhs_weights = layer.rhs_weights.data();
        kernel_layer.biases = layer.biases.data();
    }
}


template<typename dtype>
pair<Array<unsigned int>, Array<dtype>> PsiDeepT<dtype>::compile_rhs_connections_and_weights(
    const unsigned int prev_size,
    const unsigned int size,
    const unsigned int lhs_connectivity,
    const Array<unsigned int>& lhs_connections,
    const Array<dtype>& lhs_weights
) {
    const auto rhs_connectivity = size * lhs_connectivity / prev_size;

    Array<unsigned int> rhs_connections(prev_size * rhs_connectivity, this->gpu);
    Array<dtype> rhs_weights(prev_size * rhs_connectivity, this->gpu);

    vector<unsigned int> lhs_num_connections;
    lhs_num_connections.assign(prev_size, 0u);

    for(auto j = 0u; j < size; j++) {
        for(auto i = 0u; i < lhs_connectivity; i++) {
            const auto lhs_idx = lhs_connections[i * size + j];

            rhs_connections[lhs_idx * rhs_connectivity + lhs_num_connections[lhs_idx]] = j;
            rhs_weights[lhs_idx * rhs_connectivity + lhs_num_connections[lhs_idx]] = lhs_weights[
                i * size + j
            ];
            lhs_num_connections[lhs_idx]++;
        }
    }

    rhs_connections.update_device();
    rhs_weights.update_device();

    return {move(rhs_connections), move(rhs_weights)};
}


template<typename dtype>
Array<dtype> PsiDeepT<dtype>::get_params() const {
    Array<dtype> result(this->num_params, false);

    for(auto i = 0u; i < this->N; i++) {
        result[i]= get_real<dtype>(this->alpha_array[i]);
        result[this->N + i] = get_real<dtype>(this->beta_array[i]);
    }
    auto it = result.begin() + 2 * this->N;

    for(const auto& layer : this->layers) {
        copy(layer.biases.begin(), layer.biases.end(), it);
        it += layer.biases.size();
        copy(layer.lhs_weights.begin(), layer.lhs_weights.end(), it);
        it += layer.lhs_weights.size();
    }

    return result;
}


template<typename dtype>
void PsiDeepT<dtype>::set_params(const Array<dtype>& new_params) {
    for(auto i = 0u; i < this->N; i++) {
        this->alpha_array[i] = get_real<double>(new_params[i]);
        this->beta_array[i] = get_real<double>(new_params[this->N + i]);
    }
    auto it = new_params.begin() + 2 * this->N;

    for(auto layer_it = this->layers.begin(); layer_it != this->layers.end(); layer_it++) {
        auto& layer = *layer_it;

        copy(it, it + layer.biases.size(), layer.biases.begin());
        layer.biases.update_device();
        it += layer.size;

        copy(it, it + layer.lhs_weights.size(), layer.lhs_weights.begin());
        layer.lhs_weights.update_device();
        it += layer.lhs_weights.size();

        if(layer_it != this->layers.begin()) {
            prev(layer_it)->rhs_weights = this->compile_rhs_connections_and_weights(
                prev(layer_it)->size,
                layer.size,
                layer.lhs_connectivity,
                layer.lhs_connections,
                layer.lhs_weights
            ).second;
        }
    }

    this->update_kernel();
}


template struct PsiDeepT<complex_t>;
template struct PsiDeepT<double>;

} // namespace rbm_on_gpu


#endif // ENABLE_PSI_DEEP
